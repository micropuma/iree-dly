#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void add(const float* a, const float* b, float* c) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    c[i] = a[i] + b[i];
}

int main() {
    int N = 10;
    int size = N * sizeof(float);

    float h_a[N], h_b[N], h_c[N];
    for (int i = 0; i < N; i++) {
        h_a[i] = i;
        h_b[i] = i;
    }

    float *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    add<<<1, N>>>(d_a, d_b, d_c);

    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
    
    for(int i = 0; i < N; i++) {
        printf("%f + %f = %f\n", h_a[i], h_b[i], h_c[i]);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}