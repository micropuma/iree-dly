#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <hip/hip_fp16.h>
#include <mma.h>

using namespace nvcuda;

constexpr int TILE_WIDTH = 16;

__global__ void MatrixMulKernel(float *M, float *N, float *K, int width) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (row < width && col < width) {
        float Pvalue = 0;
        for (int k = 0; k < width; k++) {
            Pvalue += M[row * width + k] * N[k * width + col];
        }
        K[row * width + col] = Pvalue;
    }
}

__global__ void MatrixMulKernelTiled(float *M, float *N, float *K, int width) {
    __shared__ float d_M[TILE_WIDTH][TILE_WIDTH];
    __shared__ float d_N[TILE_WIDTH][TILE_WIDTH];

    int tx = threadIdx.x, ty = threadIdx.y;
    int bx = blockIdx.x, by = blockIdx.y;
    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;
    float Pvalue = 0.0f;
    int numTiles = (width + TILE_WIDTH - 1) / TILE_WIDTH;

    for (int ph = 0; ph < numTiles; ph++) {
        if (row < width && (ph * TILE_WIDTH + tx) < width)
            d_M[ty][tx] = M[row * width + ph * TILE_WIDTH + tx];
        else
            d_M[ty][tx] = 0.0f;

        if ((ph * TILE_WIDTH + ty) < width && col < width)
            d_N[ty][tx] = N[(ph * TILE_WIDTH + ty) * width + col];
        else
            d_N[ty][tx] = 0.0f;

        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; k++) {
            Pvalue += d_M[ty][k] * d_N[k][tx];
        }
        __syncthreads();
    }

    if (row < width && col < width) {
        K[row * width + col] = Pvalue;
    }
}

__global__ void MatrixMulKernelTensorCore(half *M, half *N, float *K, int width) {
    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a;
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::col_major> b;
    wmma::fragment<wmma::accumulator, 16, 16, 16, float> c;
    wmma::fill_fragment(c, 0.0f);

    int bx = blockIdx.x * 16;
    int by = blockIdx.y * 16;

    if (bx < width && by < width) {
        wmma::load_matrix_sync(a, M + by * width + bx, width);
        wmma::load_matrix_sync(b, N + bx * width + by, width);
        wmma::mma_sync(c, a, b, c);
        wmma::store_matrix_sync(K + by * width + bx, c, width, wmma::mem_row_major);
    }
}

void MatrixMultiplyCUDA(float *h_M, float *h_N, float *h_K, float *h_K1, float *h_K2, int width) {
    float *d_M, *d_N, *d_K;
    float *d_K1, *d_K2;
    half *d_Mh, *d_Nh;
    int size = width * width * sizeof(float);
    int half_size = width * width * sizeof(half);

    hipMalloc(&d_M, size); hipMalloc(&d_N, size); hipMalloc(&d_K, size);
    hipMalloc(&d_K1, size); hipMalloc(&d_K2, size);
    hipMalloc(&d_Mh, half_size); hipMalloc(&d_Nh, half_size);

    hipMemcpy(d_M, h_M, size, hipMemcpyHostToDevice);
    hipMemcpy(d_N, h_N, size, hipMemcpyHostToDevice);

    half *h_Mh = (half *)malloc(half_size);
    half *h_Nh = (half *)malloc(half_size);

    for (int i = 0; i < width * width; i++) {
        h_Mh[i] = __float2half(h_M[i]);
        h_Nh[i] = __float2half(h_N[i]);
    }
    
    hipMemcpy(d_Mh, h_Mh, half_size, hipMemcpyHostToDevice);
    hipMemcpy(d_Nh, h_Nh, half_size, hipMemcpyHostToDevice);

    dim3 blockDim(TILE_WIDTH, TILE_WIDTH);
    dim3 gridDim((width + TILE_WIDTH - 1) / TILE_WIDTH, (width + TILE_WIDTH - 1) / TILE_WIDTH);

    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    MatrixMulKernel<<<gridDim, blockDim>>>(d_M, d_N, d_K, width);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("MatrixMulKernel execution time: %f ms\n", time);

    hipEventRecord(start);
    MatrixMulKernelTiled<<<gridDim, blockDim>>>(d_M, d_N, d_K1, width);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("MatrixMulKernelTiled execution time: %f ms\n", time);

    hipEventRecord(start);
    MatrixMulKernelTensorCore<<<gridDim, blockDim>>>(d_Mh, d_Nh, d_K2, width);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("MatrixMulKernelTensorCore execution time: %f ms\n", time);

    hipMemcpy(h_K, d_K, size, hipMemcpyDeviceToHost);
    hipMemcpy(h_K1, d_K1, size, hipMemcpyDeviceToHost);
    hipMemcpy(h_K2, d_K2, size, hipMemcpyDeviceToHost);

    hipFree(d_M); hipFree(d_N); hipFree(d_K);
    hipFree(d_K1); hipFree(d_K2); hipFree(d_Mh); hipFree(d_Nh);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    free(h_Mh); free(h_Nh);
}

int main() {
    int width = 256;
    float *h_M = (float *)malloc(width * width * sizeof(float));
    float *h_N = (float *)malloc(width * width * sizeof(float));
    float *h_K = (float *)malloc(width * width * sizeof(float));
    float *h_K1 = (float *)malloc(width * width * sizeof(float));
    float *h_K2 = (float *)malloc(width * width * sizeof(float));

    for (int i = 0; i < width * width; i++) {
        h_M[i] = rand() % 10;
        h_N[i] = rand() % 10;
    }

    MatrixMultiplyCUDA(h_M, h_N, h_K, h_K1, h_K2, width);

    free(h_M); free(h_N); free(h_K); free(h_K1); free(h_K2);
    return 0;
}

