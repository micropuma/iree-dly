#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

// 矩阵乘法核函数
__global__ void MatrixMulKernel(float *M, float *N, float *K, int width) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < width && col < width) {
        float Pvalue = 0;
        for (int k = 0; k < width; k++) {
            Pvalue += M[row * width + k] * N[k * width + col];
        }
        K[row * width + col] = Pvalue;
    }        
}

void MatrixMultiplyCUDA(float *h_M, float *h_N, float *h_K, int width) {
    float *d_M, *d_N, *d_K;
    int size = width * width * sizeof(float);

    // 1. 设备端动态分配内存
    hipMalloc(&d_M, size);
    hipMalloc(&d_N, size);
    hipMalloc(&d_K, size);

    // 2. 复制数据到 GPU
    hipMemcpy(d_M, h_M, size, hipMemcpyHostToDevice);
    hipMemcpy(d_N, h_N, size, hipMemcpyHostToDevice);

    // 3. 计算线程块和网格大小
    dim3 blockDim(16, 16);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (width + blockDim.y - 1) / blockDim.y);

    // 4. 启动 CUDA 核函数
    MatrixMulKernel<<<gridDim, blockDim>>>(d_M, d_N, d_K, width);
    hipDeviceSynchronize();

    // 5. 复制结果回 CPU
    hipMemcpy(h_K, d_K, size, hipMemcpyDeviceToHost);

    // 6. 释放 GPU 内存
    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_K);
}

// 打印矩阵
void printMatrix(float *matrix, int width) {
    for (int i = 0; i < width; i++) {
        for (int j = 0; j < width; j++) {
            printf("%.2f ", matrix[i * width + j]);
        }
        printf("\n");
    }
}

int main() {
    int width = 4;  // 矩阵大小 (width x width)
    
    // 1. 在 CPU 端分配并初始化矩阵
    float *h_M = (float *)malloc(width * width * sizeof(float));
    float *h_N = (float *)malloc(width * width * sizeof(float));
    float *h_K = (float *)malloc(width * width * sizeof(float));

    for (int i = 0; i < width * width; i++) {
        h_M[i] = rand() % 10;  // 随机数 0~9
        h_N[i] = rand() % 10;
    }

    // 2. 调用 CUDA 进行矩阵乘法
    MatrixMultiplyCUDA(h_M, h_N, h_K, width);

    // 3. 打印结果
    printf("Matrix M:\n");
    printMatrix(h_M, width);

    printf("\nMatrix N:\n");
    printMatrix(h_N, width);

    printf("\nMatrix K (Result):\n");
    printMatrix(h_K, width);

    // 4. 释放 CPU 内存
    free(h_M);
    free(h_N);
    free(h_K);

    return 0;
}

